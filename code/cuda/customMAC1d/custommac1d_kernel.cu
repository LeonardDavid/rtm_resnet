#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <stdio.h>
#include <cstdint>

#define DEBUG_1D 0
#define DEBUG_THREAD_INFO_FLOAT32 0
#define DEBUG_THREAD_INFO_INT32 0
#define DEBUG_BITS 0
#define DEBUG_SEEDS 0

template <typename scalar_t>
__global__ void custommac1d_kernel(
    torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> input,
    torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> weight,
    torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> output,
    int array_size
  )
{

  // handle access indices
  const int c = blockIdx.x * blockDim.x + threadIdx.x; // y
  const int d = blockIdx.y * blockDim.y + threadIdx.y; // x

  // make sure we don't modify memory regions outside of output
  if ((d < output.size(0)) && (c < output.size(1)))
  {
    // this is (c,d,e), we have as many threads as we have pixels in output out
    // each thread of out calculates a MAC (row of filter times column of input)

    // every thread is responsible for one sum, there are as many threads as mac sums in output
    int cycle_counter = 0; // nr of ready inputs to majority gate
    int global_cycles = 0; // counter for all cycles
    float mac_result = 0;
    for(int i = 0; i < weight.size(1); i++)
    {
      //printf("Thread: (%d,%d,%d)\nWeight: %.4f, Input: %.4f\n", c, d, e, weight[c][i], input[d][i][e]);
      mac_result += (weight[c][i] * input[d][i]);
      // output[d][c][] += mult_result;
      cycle_counter += 1;

      // when at last element, store mac result
      if(i == (weight.size(1)-1))
      {
        output[d][c][global_cycles] = mac_result;
        mac_result = 0;
        cycle_counter = 0;
        global_cycles += 1;
      }
      else if(cycle_counter == array_size)
      {
        output[d][c][global_cycles] = mac_result;
        mac_result = 0;
        cycle_counter = 0;
        global_cycles += 1;
      }
    }
  }
}

torch::Tensor custommac1d_cuda(
  torch::Tensor input,
  torch::Tensor weight,
  torch::Tensor output,
  int array_size
) {
  // The number of thread blocks in a grid is usually dictated by the size of the data being processed, which typically exceeds the number of processors in the system.
  // dim3 threadsPerBlock(8,8,8)
  // <<<number of blocks per grid, number of threads ber block>>>
  // grid is created with enough blocks to have one thread per matrix element

  // https://devtalk.nvidia.com/default/topic/1028226/how-many-concurrent-threads-are-running-on-my-geforce-gtx-1080-ti-/
  const int output_size_x = output.size(1);
  const int output_size_y = output.size(0);
  int threads_x = 16; // per block, 16
  int threads_y = 16; // per block, 16

  #if DEBUG_1D
    threads_x = 1;
    threads_y = 1;
  #endif

  const dim3 threads(threads_x,threads_y);
  const dim3 blocks((output_size_x + threads_x - 1) / threads_x,
                    (output_size_y + threads_y - 1) / threads_y);

  AT_DISPATCH_ALL_TYPES(input.type(), "custommac1d_cuda", ([&] {
    custommac1d_kernel<scalar_t><<<blocks, threads>>>(
        input.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
        weight.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
        output.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
        array_size
    );
  }));

  return output;
}
