#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <cstdint>

// threads per block
#define TPB_X 8
#define TPB_Y 8
#define TPB_Z 8

template <typename scalar_t>
__global__ void quantize_kernel(
    torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> input,
    float min_range,
    float max_range,
    float q_range,
    int unsign
  ) {

  // float m_range = 0;
  // if (max_range > min_range)
  // {
  //   m_range = max_range;
  // }
  // else
  // {
  //   m_range = max_range;
  // }

  const int c = blockIdx.x * blockDim.x + threadIdx.x;
  const int d = blockIdx.y * blockDim.y + threadIdx.y;
  const int e = blockIdx.z * blockDim.z + threadIdx.z;

  if ((c < input.size(0)) && (d < input.size(1)) && (e < input.size(2)))
  {
    input[c][d][e] = std::round((input[c][d][e] - min_range) * q_range / (max_range - min_range));
    if (unsign == 0)
    {
      input[c][d][e] -= (q_range + 1) / 2.0;
    }
    // input[c][d][e] /= q_range;
    // input[c][d][e] *= m_range;
  }
}

torch::Tensor quantize_cuda(
  torch::Tensor input,
  float min_range,
  float max_range,
  int q,
  int unsign
) {

  int64_t shape_len = input.dim();
  std::vector<int64_t> shape_original;
  for (int i = 0; i < shape_len; i++)
  {
    shape_original.push_back(input.size(i));
  }

  if (shape_len == 1)
  {
    input = input.reshape({input.size(0),1,1});
  }
  if (shape_len == 2)
  {
    input = input.reshape({input.size(0),input.size(1),1});
  }
  if (shape_len > 3)
  {
    input = input.reshape({input.size(0),input.size(1),-1});
  }
  // The number of thread blocks in a grid is usually dictated by the size of the data being processed, which typically exceeds the number of processors in the system.
  // dim3 threadsPerBlock(8,8,8)
  // <<<number of blocks per grid, number of threads ber block>>>
  // grid is created with enough blocks to have one thread per matrix element

  // https://devtalk.nvidia.com/default/topic/1028226/how-many-concurrent-threads-are-running-on-my-geforce-gtx-1080-ti-/
  const int input_size_x = input.size(0);
  const int input_size_y = input.size(1);
  const int input_size_z = input.size(2);
  int threads_x = TPB_X; // per block, 8
  int threads_y = TPB_Y;
  int threads_z = TPB_Z;

  const float q_range = std::pow(2, q) - 1.0;

  const dim3 threads(threads_x,threads_y, threads_z);
  const dim3 blocks((input_size_x + threads_x - 1) / threads_x,
                    (input_size_y + threads_y - 1) / threads_y,
                    (input_size_z + threads_z - 1) / threads_z);

  AT_DISPATCH_ALL_TYPES(input.type(), "quantize_cuda", ([&] {
    quantize_kernel<scalar_t><<<blocks, threads>>>(
        input.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
        min_range,
        max_range,
        q_range,
        unsign
    );
  }));

  input = input.reshape(shape_original);
  return input;
}
